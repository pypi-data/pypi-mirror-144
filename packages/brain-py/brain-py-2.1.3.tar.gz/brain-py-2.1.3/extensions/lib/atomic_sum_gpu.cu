#include "hip/hip_runtime.h"
// This file contains the GPU implementation of our op. It's a pretty typical CUDA kernel
// and I make no promises about the quality of the code or the choices made therein, but
// it should get the point across.

#include "atomic_sum_gpu.h"

namespace brainpy_lib {

    namespace {

// "atomic_sum" operator //
        template<typename F, typename I>
        __global__ void gpu_atomic_sum_homo_kernel(const std::uint32_t size,
                                                   const F &value,
                                                   const I *post_ids,
                                                   F *result) {
            for (std::uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
                 i < size; i += blockDim.x * gridDim.x) {
                atomicAdd(&result[post_ids[i]], value);
            }
        }

        template<typename F, typename I>
        inline void gpu_atomic_sum_homo(hipStream_t stream,
                                        void **buffers,
                                        const char *opaque,
                                        std::size_t opaque_len) {
            // size
            const AtomicSumDescriptor &d = *UnpackDescriptor<AtomicSumDescriptor>(opaque, opaque_len);
            const std::uint32_t conn_size = d.conn_size;
            const std::uint32_t post_size = d.post_size;

            // input and output data
            const F *values = reinterpret_cast<const F *>(buffers[0]);  // scalar as a vector
            const I *post_ids = reinterpret_cast<const I *>(buffers[1]);
            F *result = reinterpret_cast<F *>(buffers[2]);

            // call kernel
            const int block_dim = 512;
            const int grid_dim = std::min<int>(1024, (conn_size + block_dim - 1) / block_dim);
            hipMemset(result, 0, sizeof(F) * post_size);
            gpu_atomic_sum_homo_kernel<F, I><<<grid_dim, block_dim, 0, stream>>>(conn_size, values[0], post_ids,
                                                                                 result);
            ThrowIfError(hipGetLastError());
        }

        template<typename F, typename I>
        __global__ void gpu_atomic_sum_heter_kernel(const std::uint32_t size,
                                                    const F *values,
                                                    const I *post_ids,
                                                    const I *pre_ids,
                                                    F *result) {
            for (std::uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
                 i < size; i += blockDim.x * gridDim.x) {
                atomicAdd(&result[post_ids[i]], values[pre_ids[i]]);
            }
        }

        template<typename F, typename I>
        inline void gpu_atomic_sum_heter(hipStream_t stream,
                                         void **buffers,
                                         const char *opaque,
                                         std::size_t opaque_len) {
            // size
            const AtomicSumDescriptor &d = *UnpackDescriptor<AtomicSumDescriptor>(opaque, opaque_len);
            const std::uint32_t conn_size = d.conn_size;
            const std::uint32_t post_size = d.post_size;

            // input and output data
            const F *values = reinterpret_cast<const F *>(buffers[0]);  // scalar as a vector
            const I *post_ids = reinterpret_cast<const I *>(buffers[1]);
            const I *pre_ids = reinterpret_cast<const I *>(buffers[2]);
            F *result = reinterpret_cast<F *>(buffers[3]);

            // call kernel
            const int block_dim = 512;
            const int grid_dim = std::min<int>(1024, (conn_size + block_dim - 1) / block_dim);
            hipMemset(result, 0, sizeof(F) * post_size);
            gpu_atomic_sum_heter_kernel<F, I><<<grid_dim, block_dim, 0, stream>>>(conn_size, values, post_ids, pre_ids,
                                                                                  result);
            ThrowIfError(hipGetLastError());
        }


    }  // namespace


// Descriptor
    pybind11::bytes build_atomic_sum_descriptor(std::uint32_t conn_size,
                                                std::uint32_t post_size) {
        return PackDescriptor(AtomicSumDescriptor{conn_size, post_size});
    }

// homogenous atomic sum
    void gpu_atomic_sum_homo_f32_i32(hipStream_t stream,
                                     void **buffers,
                                     const char *opaque,
                                     std::size_t opaque_len) {
        gpu_atomic_sum_homo<float, std::uint32_t>(stream, buffers, opaque, opaque_len);
    }

    void gpu_atomic_sum_homo_f32_i64(hipStream_t stream,
                                     void **buffers,
                                     const char *opaque,
                                     std::size_t opaque_len) {
        gpu_atomic_sum_homo<float, std::uint64_t>(stream, buffers, opaque, opaque_len);
    }

    void gpu_atomic_sum_homo_f64_i32(hipStream_t stream,
                                     void **buffers,
                                     const char *opaque,
                                     std::size_t opaque_len) {
        gpu_atomic_sum_homo<double, std::uint32_t>(stream, buffers, opaque, opaque_len);
    }

    void gpu_atomic_sum_homo_f64_i64(hipStream_t stream,
                                     void **buffers,
                                     const char *opaque,
                                     std::size_t opaque_len) {
        gpu_atomic_sum_homo<double, std::uint64_t>(stream, buffers, opaque, opaque_len);
    }

// heterogeneous atomic sum
    void gpu_atomic_sum_heter_f32_i32(hipStream_t stream,
                                      void **buffers,
                                      const char *opaque,
                                      std::size_t opaque_len) {
        gpu_atomic_sum_heter<float, std::uint32_t>(stream, buffers, opaque, opaque_len);
    }

    void gpu_atomic_sum_heter_f32_i64(hipStream_t stream,
                                      void **buffers,
                                      const char *opaque,
                                      std::size_t opaque_len) {
        gpu_atomic_sum_heter<float, std::uint64_t>(stream, buffers, opaque, opaque_len);
    }

    void gpu_atomic_sum_heter_f64_i32(hipStream_t stream,
                                      void **buffers,
                                      const char *opaque,
                                      std::size_t opaque_len) {
        gpu_atomic_sum_heter<double, std::uint32_t>(stream, buffers, opaque, opaque_len);
    }

    void gpu_atomic_sum_heter_f64_i64(hipStream_t stream,
                                      void **buffers,
                                      const char *opaque,
                                      std::size_t opaque_len) {
        gpu_atomic_sum_heter<double, std::uint64_t>(stream, buffers, opaque, opaque_len);
    }


}  // namespace brainpylib
